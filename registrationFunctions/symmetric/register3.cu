#include <hip/hip_runtime.h>

__global__ void interp3(double *imgout, 
	double *f_c, 
	double *f_r, 
	double *f_b,
	double *imgin, 
	int rows, 
	int cols, 
	int beams)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y % (rows/blockDim.y) * blockDim.y + threadIdx.y;
	int k = blockIdx.y / (rows/blockDim.y) * blockDim.z + threadIdx.z;

	if (i >= cols - 1 || j >= rows - 1 || k >= beams - 1)
		return;

	double i_o_f = f_c[i*rows + j + k*rows*cols] - 1;
	double j_o_f = f_r[i*rows + j + k*rows*cols] - 1;
	double k_o_f = f_b[i*rows + j + k*rows*cols] - 1;

	//first along i//
	int j_d = (int) j_o_f;
	int k_d = (int) k_o_f;
	double w_d = floor(i_o_f + 1) - i_o_f;
	double w_u = 1.0f - w_d;

	double R00 = w_d * imgin[((int) i_o_f) * rows + j_d + k_d*rows*cols] + w_u * imgin[((int) (i_o_f + 1))*rows + j_d + k_d*rows*cols];
	double R10 = w_d * imgin[((int) i_o_f) * rows + (j_d + 1) + k_d*rows*cols] + w_u * imgin[((int) (i_o_f + 1))*rows + (j_d + 1) + k_d*rows*cols];
	double R01 = w_d * imgin[((int) i_o_f) * rows + j_d + (k_d + 1)*rows*cols] + w_u * imgin[((int) (i_o_f + 1))*rows + j_d + (k_d + 1)*rows*cols];
	double R11 = w_d * imgin[((int) i_o_f) * rows + (j_d + 1) + (k_d + 1)*rows*cols] + w_u * imgin[((int) (i_o_f + 1))*rows + (j_d + 1) + (k_d + 1)*rows*cols];

	//now along j//
	w_d = floor(j_o_f + 1) - j_o_f;
	w_u = 1.0f - w_d;

	double R0 = w_d * R00 + w_u * R10;
	double R1 = w_d * R01 + w_u * R11;

	//finally along k//
	w_d = floor(k_o_f + 1) - k_o_f;
	w_u = 1.0f - w_d;

	imgout[i*rows + j + k*cols*rows] = w_d * R0 + w_u * R1;

}	

__global__ void extf (
	double *out,
	double *img1,
	double *img2,
	double *grad,
	int rows,
	int cols,
	int beams)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y % (rows/blockDim.y) * blockDim.y + threadIdx.y;
	int k = blockIdx.y / (rows/blockDim.y) * blockDim.z + threadIdx.z;

	if (i >= cols || j >= rows || k >= beams)
		return;

	out[i*rows + j + k*rows*cols] = (img1[i*rows + j + k*rows*cols]
		- img2[i*rows + j + k*rows*cols])
		* grad[i*rows + j + k*rows*cols];

}

__global__ void intf(
	double *out,
	double *f,
	int rows,
	int cols,
	int beams)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y % (rows/blockDim.y) * blockDim.y + threadIdx.y;
	int k = blockIdx.y / (rows/blockDim.y) * blockDim.z + threadIdx.z;

	if (i >= cols - 1 || j >= rows - 1 || k >= beams - 1 || i < 1 || j < 1 || k < 1)
		return;

	out[i*rows + j + k*rows*cols] = -6 * f[i*rows + j + k*rows*cols] +
		f[(i-1)*rows + j + k*rows*cols] + f[(i+1)*rows + j + k*rows*cols] +
		f[i*rows + (j-1) + k*rows*cols] + f[i*rows + (j+1) + k*rows*cols] +
		f[i*rows + j + (k-1)*rows*cols] + f[i*rows + j + (k+1)*rows*cols];
}

__global__ void d_f(
	double *out,
	double *extf,
	double *intf,
	double rho,
	double lambda,
	int rows,
	int cols,
	int beams)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y % (rows/blockDim.y) * blockDim.y + threadIdx.y;
	int k = blockIdx.y / (rows/blockDim.y) * blockDim.z + threadIdx.z;

	if (i >= cols || j >= rows || k >= beams)
		return;

	out[i*rows + j + k*rows*cols] = rho * 
		(extf[i*rows + j + k*rows*cols] + lambda*intf[i*rows + j + k*rows*cols]);
}