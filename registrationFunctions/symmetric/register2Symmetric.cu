#include <hip/hip_runtime.h>

__global__ void interp2(double *imgout, double *fCol, double *fRow, double *imgin, int rows, int cols)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i >= rows || j >= cols)
		return;

	double i_o_f = fCol[i * cols + j];
	double j_o_f = fRow[i * cols + j];

	i_o_f = fmax(1.0, fmin(i_o_f, (double) cols));
	j_o_f = fmax(1.0, fmin(j_o_f, (double) rows));

	//we will interpolate x direction first, giving R1 and R2//
	double R1 = (floor(i_o_f + 1) - i_o_f) * imgin[(int) floor(i_o_f - 1) * cols + (int) floor(j_o_f - 1)] + (i_o_f - floor(i_o_f)) * imgin[(int) ceil(i_o_f - 1) * cols + (int) floor(j_o_f - 1)];
	double R2 = (floor(i_o_f + 1) - i_o_f) * imgin[(int) floor(i_o_f - 1) * cols + (int) ceil(j_o_f - 1)] + (i_o_f - floor(i_o_f)) * imgin[(int) ceil(i_o_f - 1) * cols + (int) ceil(j_o_f - 1)];

	//now finish//
	imgout[i * cols + j] = (floor(j_o_f + 1) - j_o_f) * R1 + (j_o_f - floor(j_o_f)) * R2;
}

__global__ void extf( double *out, double *img1, double *img2, double *grad, int rows, int columns )
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i >= columns || j >= rows)
		return;
	
	out[i + j * columns] = 2 * (img1[i + j * columns] - img2[i + j * columns]) * grad[i + j * columns];
	
}

__global__ void jacPartialsAndBarrier(
	double *i_m_1,
	double *i_p_1,
	double *j_m_1,
	double *j_p_1,
	double *barrier,
	double *jac,
	double *f,
	double *img1,
	double *img2,
	int rows,
	int columns,
	int flip) //-1 for F_y, 1 for F_x// 
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;//to be consistent with matlab indexing//
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i < 1 || j < 1 || i  >= (columns - 1) || j >= (rows - 1))
		return;

	i_m_1[i * rows + j] = flip * .25 * (f[(i - 1) * rows + (j + 1)] - f[(i - 1) * rows + (j - 1)]);
	i_p_1[i * rows + j] = flip * .25 * (f[(i + 1) * rows + (j - 1)] - f[(i + 1) * rows + (j + 1)]);
	j_m_1[i * rows + j] = flip * .25 * (f[(i - 1) * rows + (j - 1)] - f[(i + 1) * rows + (j - 1)]);
	j_p_1[i * rows + j] = flip * .25 * (f[(i + 1) * rows + (j + 1)] - f[(i - 1) * rows + (j + 1)]);

	//barrier//
	barrier[i * rows + j] = -1 * (
		i_m_1[i * rows + j] * (log(jac[(i - 1) * rows + j]) - 1) / pow(jac[(i - 1) * rows + j], 2) +
		i_p_1[i * rows + j] * (log(jac[(i + 1) * rows + j]) - 1) / pow(jac[(i + 1) * rows + j], 2) +
		j_m_1[i * rows + j] * (log(jac[i * rows + (j - 1)]) - 1) / pow(jac[i * rows + (j - 1)], 2) +
		j_p_1[i * rows + j] * (log(jac[i * rows + (j + 1)]) - 1) / pow(jac[i * rows + (j + 1)], 2));

	//multiply the partials by the image difference//
	i_m_1[i * rows + j] *= pow(img1[(i - 1) * rows + j] - img2[(i - 1) * rows + j], 2);
	i_p_1[i * rows + j] *= pow(img1[(i + 1) * rows + j] - img2[(i + 1) * rows + j], 2);
	j_m_1[i * rows + j] *= pow(img1[i * rows + (j - 1)] - img2[i * rows + (j - 1)], 2);
	j_p_1[i * rows + j] *= pow(img1[i * rows + (j + 1)] - img2[i * rows + (j + 1)], 2);
}

__global__ void intf(
	double *out,
	double *f,
	int rows,
	int columns)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i < 1 || j < 1 || i  >= (columns - 1) || j >= (rows - 1))
		return;

	out[i * rows + j] = -4 * f[i * rows + j] + f[(i - 1) * rows + j] + f[(i + 1) * rows +j] + f[i * rows + (j - 1)] + f[i * rows + (j+ 1)];
}

__global__ void jacobian(
	double *out,
	double *f_c,
	double *f_r,
	int rows,
	int columns)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i < 1 || j < 1 || i  >= (columns - 1) || j >= (rows - 1))
		return;

	out[i * rows + j] = .25 *
		((f_c[(i + 1) * rows + j] - f_c[(i - 1) * rows + j]) * (f_r[i * rows + (j + 1)] - f_r[i * rows + (j - 1)]) -
		(f_c[i * rows + (j + 1)] - f_c[i * rows + (j - 1)]) * (f_r[(i + 1) * rows + j] - f_r[(i - 1) * rows + j])); 
}

__global__ void add(
	double *out,
	double *in1,
	double *in2,
	int rows,
	int columns)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i < 2 || j < 2 || i  >= (columns - 2) || j >= (rows - 2))
		return;

	out[i * rows + j] = in1[i * rows + j] + in2[i * rows + j];
}


__global__ void d_f(
	double *out,
	double *jacf,
	double *jacg,
	double *i_m_1,
	double *j_m_1,
	double *i_p_1,
	double *j_p_1,
	double *barrier,
	double *intf,
	double *extf,
	double rho,
	double lambda,
	double lambda2,
	int rows,
	int columns)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	// if (i < 1 || j < 1 || i  >= (columns - 1) || j >= (rows - 1))
	// 	return;

	out[i * rows + j] = rho * (extf[i * rows + j] * (jacf[i * rows + j] + jacg[i * rows + j]) +
		i_m_1[i * rows + j] + i_p_1[i * rows + j] + j_m_1[i * rows + j] + j_p_1[i * rows + j] +
		lambda * intf[i * rows + j] + lambda2 * barrier[i * rows + j]);
}