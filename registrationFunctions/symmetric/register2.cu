#include <hip/hip_runtime.h>

__global__ void interp2(double *imgout, double *fCol, double *fRow, double *imgin, int rows, int cols)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i >= rows || j >= cols)
		return;

	double i_o_f = fCol[i * cols + j];
	double j_o_f = fRow[i * cols + j];

	i_o_f = fmax(1.0, fmin(i_o_f, (double) cols));
	j_o_f = fmax(1.0, fmin(j_o_f, (double) rows));

	//we will interpolate x direction first, giving R1 and R2//
	double R1 = (floor(i_o_f + 1) - i_o_f) * imgin[(int) floor(i_o_f - 1) * cols + (int) floor(j_o_f - 1)] + (i_o_f - floor(i_o_f)) * imgin[(int) ceil(i_o_f - 1) * cols + (int) floor(j_o_f - 1)];
	double R2 = (floor(i_o_f + 1) - i_o_f) * imgin[(int) floor(i_o_f - 1) * cols + (int) ceil(j_o_f - 1)] + (i_o_f - floor(i_o_f)) * imgin[(int) ceil(i_o_f - 1) * cols + (int) ceil(j_o_f - 1)];

	//now finish//
	imgout[i * cols + j] = (floor(j_o_f + 1) - j_o_f) * R1 + (j_o_f - floor(j_o_f)) * R2;
}

__global__ void extf( double *out, double *img1, double *img2, double *grad, int rows, int columns )
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	// if (i >= columns || j >= rows)
	// 	return;
	
	out[i + j * columns] = 2 * (img1[i + j * columns] - img2[i + j * columns]) * grad[i + j * columns];
	
}

__global__ void intf(
	double *out,
	double *f,
	int rows,
	int columns)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i < 1 || j < 1 || i  >= (columns - 1) || j >= (rows - 1))
		return;

	out[i * rows + j] = -4 * f[i * rows + j] + f[(i - 1) * rows + j] + f[(i + 1) * rows +j] + f[i * rows + (j - 1)] + f[i * rows + (j+ 1)];
}

__global__ void add(
	double *out,
	double *in1,
	double *in2,
	int rows,
	int columns)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i < 2 || j < 2 || i  >= (columns - 2) || j >= (rows - 2))
		return;

	out[i * rows + j] = in1[i * rows + j] + in2[i * rows + j];
}


__global__ void d_f(
	double *out,
	double *intf,
	double *extf,
	double rho,
	double lambda,
	int rows,
	int columns)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i < 1 || j < 1 || i  >= (columns - 1) || j >= (rows - 1))
		return;

	out[i * rows + j] = rho * (extf[i * rows + j] + lambda * intf[i * rows + j]);
}